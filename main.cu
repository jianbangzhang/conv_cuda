
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <climits>
#include <cstdlib>
#include <endian.h>
#include <sys/wait.h>

static void HandleError(hipError_t err,
    const char *file,int line){
  if (err!=hipSuccess){
    printf("%s in %s at line %d\n",hipGetErrorString(err),file,line);
    exit(EXIT_FAILURE);
  }
}

#define CHECK(error){HandleError(error,__FILE__,__LINE__);}

int getThreadNum(){
  hipDeviceProp_t prop;
  int count;

  CHECK(hipGetDeviceCount(&count));
  printf("gpu num %d\n",count);
  CHECK(hipGetDeviceProperties(&prop, 0));
  printf("max thread num: %d\n",prop.maxThreadsPerBlock);
  printf("max grid dimemsions: %d, %d,%d)\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
  return prop.maxThreadsPerBlock;
}


__global__ void conv(float *img,float *kernel,float *result,const int width,const int height,const int kernel_size){
  int ti=threadIdx.x;
  int bi=blockIdx.x;
  int id=(bi*blockDim.x+ti);
  if (id>=width*height){
    return;
  }

  int row=id/width;
  int col=id%width;

  for (int i=0;i<kernel_size;++i){
    for (int j=0;j<kernel_size;++j){
      float img_value=0;
      int curRow=row-kernel_size/2+i;
      int curCol=col-kernel_size/2+j;
      if (curRow<0||curCol<0||curRow>=height||curCol>=width){
      }else{
        img_value=img[curRow*width+curCol];
      }
      result[id]+=kernel[i*kernel_size+j]*img_value;
    }
  }
}

int main(void){
  const int width=1920;
  const int height=1080;
  float *img=new float[width*height];
  float *result=new float[width*height];
  for (int row=0;row<height;++row){
    for (int col=0;col<width;++col){
      img[col+row*width]=(col+row)%256;
    }
  }

  const int kernel_size=3;
  float *kernel=new float[kernel_size*kernel_size];
  for (int i=0;i<kernel_size*kernel_size;++i){
    kernel[i]=i%kernel_size-1;
  }
  //visualization
  
  for (int row=0;row<10;++row){
    for (int col=0;col<10;++col){
      printf("%2.0f ",img[col+row*width]);
    }
    printf("\n");
  }
  printf("kernel\n");
  for (int row=0;row<kernel_size;++row){
    for (int col=0;col<kernel_size;++col){
      printf("%2.0f ",kernel[col+row*kernel_size]);
    }
    printf("\n");
  }

  float *img_gpu;
  float *kernel_gpu;
  float *result_gpu;

  CHECK(hipMalloc((void**)&img_gpu,width*height*sizeof(float)));
  CHECK(hipMalloc((void**)&kernel_gpu,kernel_size*kernel_size*sizeof(float)));
  CHECK(hipMalloc((void**)&result_gpu,width*height*sizeof(float)));
  CHECK(hipMemcpy(img_gpu,img,width*height*sizeof(float),hipMemcpyHostToDevice));
  CHECK(hipMemcpy(kernel_gpu,kernel,kernel_size*kernel_size*sizeof(float),hipMemcpyHostToDevice));

  int threadNum=getThreadNum();
  int blockNum=(width*height-0.5)/threadNum+1;

  conv<<<blockNum,threadNum >>>(img_gpu,kernel_gpu,result_gpu,width,height,kernel_size);
  CHECK(hipMemcpy(result,result_gpu,width*height*sizeof(float),hipMemcpyDeviceToHost));

  printf("result\n");
  for (int row=0;row<10;++row){
    for (int col=0;col<10;++col){
      printf("%2.0f ",result[col+row*width]);
    }
    printf("\n");
  }
  return 0;
}
